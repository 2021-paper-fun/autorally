#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
 * @file generalized_linear.cu
 * @author Grady Williams <gradyrw@gmail.com>
 * @date May 24, 2017
 * @copyright 2017 Georgia Institute of Technology
 * @brief Class implementation for generalized linear models
 ***********************************************/

namespace autorally_control {

#include "gpu_err_chk.h"

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::GeneralizedLinear(
                    Eigen::Matrix<float, DYNAMICS_DIM, NUM_BFS, Eigen::RowMajor> theta, 
                    float delta_t, float2* control_rngs)
{
  int i;
  dt_ = delta_t;
  basisFunctions_ = new BF();
  kinematics_ = new K_FUNC();
  if (control_rngs == NULL){
    control_rngs_ = new float2[CONTROL_DIM];
    for (i = 0; i < CONTROL_DIM; i++){
      control_rngs_[i].x = -FLT_MAX;
      control_rngs_[i].y = FLT_MAX;
    }
  }
  else {
    control_rngs_ = control_rngs;
  }
  HANDLE_ERROR( hipMalloc((void**)&control_rngs_d_, CONTROL_DIM*sizeof(float2)) );
  HANDLE_ERROR( hipMalloc((void**)&theta_d_, NUM_BFS*DYNAMICS_DIM*sizeof(float)) );
  setParams(theta);
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::GeneralizedLinear(float delta_t, float2* control_rngs)
{
  int i;
  dt_ = delta_t;
  basisFunctions_ = new BF();
  kinematics_ = new K_FUNC();
  if (control_rngs == NULL){
    control_rngs_ = new float2[CONTROL_DIM];
    for (i = 0; i < CONTROL_DIM; i++){
      control_rngs_[i].x = -FLT_MAX;
      control_rngs_[i].y = FLT_MAX;
    }
  }
  else {
    control_rngs_ = control_rngs;
  }
  HANDLE_ERROR( hipMalloc((void**)&control_rngs_d_, CONTROL_DIM*sizeof(float2)) );
  HANDLE_ERROR( hipMalloc((void**)&theta_d_, NUM_BFS*DYNAMICS_DIM*sizeof(float)) );
  HANDLE_ERROR( hipMemcpy(control_rngs_d_, control_rngs_, CONTROL_DIM*sizeof(float2), hipMemcpyHostToDevice) );
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::setParams(Eigen::Matrix<float, DYNAMICS_DIM, NUM_BFS, Eigen::RowMajor> theta)
{
  theta_ = theta;
  paramsToDevice(); //Save to GPU memory
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::loadParams(std::string model_path)
{
  int i,j;
  if (!fileExists(model_path)){
    ROS_FATAL("Could not load generalized linear model at path: %s", model_path.c_str());
  }
  Eigen::Matrix<float, DYNAMICS_DIM, NUM_BFS, Eigen::RowMajor> theta;
  cnpy::npz_t param_dict = cnpy::npz_load(model_path);
  cnpy::NpyArray theta_raw = param_dict["W"];
  double* theta_ptr = theta_raw.data<double>();
  for (i = 0; i < DYNAMICS_DIM; i++){
    for (j = 0; j < NUM_BFS; j++){
      theta(i,j) = (float)theta_ptr[i*NUM_BFS + j];
    }
  }
  setParams(theta);
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::paramsToDevice()
{
  //Transfer CPU params to the GPU
  HANDLE_ERROR( hipMemcpy(theta_d_, theta_.data(), NUM_BFS*DYNAMICS_DIM*sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(control_rngs_d_, control_rngs_, CONTROL_DIM*sizeof(float2), hipMemcpyHostToDevice) );
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::freeCudaMem()
{
  HANDLE_ERROR(hipFree(theta_d_));
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::enforceConstraints(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  int i;
  for (i = 0; i < CONTROL_DIM; i++){
    if (control(i) < control_rngs_[i].x){
      control(i) = control_rngs_[i].x;
    }
    else if (control(i) > control_rngs_[i].y){
      control(i) = control_rngs_[i].y;
    }
  }
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::updateState(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  enforceConstraints(state, control);
  computeKinematics(state);
  computeDynamics(state, control);
  state += state_der_*dt_;
  state_der_ *= 0;
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::computeKinematics(Eigen::MatrixXf &state)
{
  state_der_(0) = cosf(state(2))*state(4) - sinf(state(2))*state(5);
  state_der_(1) = sinf(state(2))*state(4) + cosf(state(2))*state(5);
  state_der_(2) = -state(6); //Pose estimate actually gives the negative yaw derivative
}


template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::computeDynamics(Eigen::MatrixXf &state, Eigen::MatrixXf &control)
{
  /*int i;
  //Now compute the basis functions.
  for (i = 0; i < NUM_BFS; i++){
    bf_vec_(i) = basisFunctions_->basisFuncX(i, state.data(), control.data());
  }
  state_der_.block(STATE_DIM - DYNAMICS_DIM, 0, DYNAMICS_DIM, 1) = theta_*bf_vec_;*/
float para[] =  {1.33883129559920,0.391701415322727,0.0950009708432143,2.95361615969189,0.828188220600163,0.957693850834835,-0.329780228856469,-0.0434960411161758,0.198419237769060,0.248873766728441,0.00981370687408516,7.79994311191890,21.1381472017746}; 
    float L_axis = 0.57;
    float g = 9.81;  //[kg*m/s^2]
    float r_F=0.095; // front wheel radius [m]
    float r_R=0.095; //rear wheel radius [m]
     
    float Iz=para[0];  // mass moment of inertia of vehicle around z-axis
    float lf=para[1];  // distance from mass center to front axis [m]
    float lr=L_axis-lf;  // distance from mass center to rear axis [m]
    float h=para[2];  // height of mass center [m]
     
    float B=para[3];
    float C=para[4]; 
    float D=para[5]; 
    float E=para[6];
    float Sh=para[7];
    float Sv=para[8];  // B - Sv  parameters in tire force model
    
    float scl=para[9];
    float Iwr=para[10];   // mass moment of inertia of rear wheel
    float Gear=para[11];  // gear ratio of the transmission
    float m=para[12];   // vehicle mass [kg]

    float delta = -scl*control(0);
    float Tr = Gear*control(1);

    float Vx = state(4);
    float Vy = state(5);
    float r = -state(6);
    float wR = state(7);
    
    float V_fx = Vx;
    float V_fy = Vy+r*lf;
    float V_rx = Vx;
    float V_ry = Vy-r*lr;

    float VFx = V_fx*cosf(delta)+V_fy*sinf(delta);
    float VFy = -V_fx*sinf(delta)+V_fy*cosf(delta);
    float VRx = V_rx;
    float VRy = V_ry;

    std::cout<<para;
 
    if (wR*r_R<0.05){ 
       state_der_(3) = 0;
       state_der_(5) = 0;
       state_der_(6) = 0;
       state_der_(7) = Tr/Iwr;
       state_der_(4) = 0.8*state_der_(7)*r_R;
	}
 
    else {
       float sFx = 0;
       float sFy = VFy/VFx;
       float sF = sqrtf( sFx*sFx+sFy*sFy );
       float sRx = ( VRx-wR*r_R )/( wR*r_R );
       float sRy = VRy/( wR*r_R ); 
       float sR = sqrtf( sRx*sRx+sRy*sRy );
       float mu_sF=D*sinf( C*atanf( B*( (1-E)*(sF+Sh)+E/B*atanf(B*(sF+Sh)) ) ) ) +Sv;
       float mu_sR=D*sinf( C*atanf( B*( (1-E)*(sR+Sh)+E/B*atanf(B*(sR+Sh)) ) ) ) +Sv;
       
       float fFx,fFy,fRx,fRy;
       if (sF==0){
//	  float fFx=0;
//	  float fFy=0; 

	  fFx=0;
	  fFy=0; 	     
	  float muRx=-sRx*mu_sR/sR;
	  float muRy=-sRy*mu_sR/sR;

          float fFz=( lr-h*muRx )*m*g/( L_axis-h*muRx );
          float fRz=m*g-fFz;
 
//          float fRx=muRx*fRz;
//          float fRy=muRy*fRz;
          fRx=muRx*fRz;
          fRy=muRy*fRz;        
        }
       else {
	  float muFx=-sFx*mu_sF/sF;
	  float muFy=-sFy*mu_sF/sF;  
	  float muRx=-sRx*mu_sR/sR;
	  float muRy=-sRy*mu_sR/sR; 

          float fFz=( lr-h*muRx )*m*g/( L_axis+h*( muFx*cosf( delta )-muFy*sinf( delta )- muRx) );
          float fRz=m*g-fFz;
 
          fFx=muFx*fFz;
          fFy=muFy*fFz;
          fRx=muRx*fRz;
          fRy=muRy*fRz;       
        }

        state_der_(3) = 0;
	state_der_(4) = ( fFx*cosf( delta )-fFy*sinf( delta )+fRx )/m+Vy*r;
        state_der_(5) = ( fFx*sinf( delta )+fFy*cosf( delta )+fRy )/m-Vx*r;
        state_der_(6) = -( ( fFy*cosf( delta )+fFx*sinf( delta ) )*lf-fRy*lr )/Iz;
        state_der_(7) = (Tr-fRx*r_R)/Iwr;

        }

}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
__device__ void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::cudaInit(float* theta_s)
{
  //Transfers the global memory cuda data to a shared memory array.
  int i,j;
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  for (i = tdy; i < DYNAMICS_DIM; i+= blockDim.y) {
    for (j = tdx; j < NUM_BFS; j+= blockDim.x) {
      theta_s[i*NUM_BFS + j] = theta_d_[i*NUM_BFS + j];
    }
  }
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
__device__ void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::enforceConstraints(float* state, float* control)
{
  int i;
  for (i = 0; i < CONTROL_DIM; i++){
    if (control[i] < control_rngs_d_[i].x){
      control[i] = control_rngs_d_[i].x;
    }
    else if (control[i] > control_rngs_d_[i].y){
      control[i] = control_rngs_d_[i].y;
    }
  }
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
__device__ void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::computeStateDeriv(float* s, float* u, float* s_der, float* theta_s)
{
  if (threadIdx.y == 0){
    computeKinematics(s, s_der);
  }
  computeDynamics(s, u, s_der, theta_s);
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
__device__ void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::incrementState(float* state, float* state_der)
{
  int i;
  int tdy = threadIdx.y;
  //Add the state derivative time dt to the current state.
  for (i = tdy; i < STATE_DIM; i+=blockDim.y){
    state[i] += state_der[i]*dt_;
    state_der[i] = 0; //Important: reset the state derivative to zero.
  }
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
__device__ void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::computeKinematics(float* state, float* state_der)
{
  state_der[0] = cosf(state[2])*state[4] - sinf(state[2])*state[5];
  state_der[1] = sinf(state[2])*state[4] + cosf(state[2])*state[5];
  state_der[2] = -state[6]; //Pose estimate actually gives the negative yaw derivative
}

template<class BF, int S_DIM, int C_DIM, int BF_DIM, class K_FUNC, int K_DIM>
__device__ void GeneralizedLinear<BF, S_DIM, C_DIM, BF_DIM, K_FUNC, K_DIM>::computeDynamics(float* s, float* u, float* s_der, float* theta_s)
{
  /*int i,j;
  int tdy = threadIdx.y;
  float bf_temp; //Temporary variable for storing basis function evaluations.
  float eval_temp[DYNAMICS_DIM]; //Temporary variable to reduce the number of atomic adds.
  for (i = 0; i < DYNAMICS_DIM; i++){
    eval_temp[i] = 0;
  }
  for (i = tdy; i < NUM_BFS; i += blockDim.y) {
    bf_temp = basisFunctions_->basisFuncX(i, s, u);
    for (j = 0; j < DYNAMICS_DIM; j++) {
      eval_temp[j] += theta_s[j*NUM_BFS + i]*bf_temp;
    }
  }
  //Add to state using atomic add.
  for (i = 0; i < DYNAMICS_DIM; i++){
    atomicAdd(&s_der[i+(STATE_DIM - DYNAMICS_DIM)], eval_temp[i]);
  }*/
float para[] = {1.33883129559920,0.391701415322727,0.0950009708432143,2.95361615969189,0.828188220600163,0.957693850834835,-0.329780228856469,-0.0434960411161758,0.198419237769060,0.248873766728441,0.00981370687408516,7.79994311191890,21.1381472017746}; 
    float L_axis = 0.57;
    float g = 9.81;  //[kg*m/s^2]
    float r_F=0.095; // front wheel radius [m]
    float r_R=0.095; //rear wheel radius [m]
     
    float Iz=para[0];  // mass moment of inertia of vehicle around z-axis
    float lf=para[1];  // distance from mass center to front axis [m]
    float lr=L_axis-lf;  // distance from mass center to rear axis [m]
    float h=para[2];  // height of mass center [m]
     
    float B=para[3];
    float C=para[4]; 
    float D=para[5]; 
    float E=para[6];
    float Sh=para[7];
    float Sv=para[8];  // B - Sv  parameters in tire force model
    
    float scl=para[9];
    float Iwr=para[10];   // mass moment of inertia of rear wheel
    float Gear=para[11];  // gear ratio of the transmission
    float m=para[12];   // vehicle mass [kg]

    float delta = -scl*u[0];
    float Tr = Gear*u[1];

    float Vx = s[4];
    float Vy = s[5];
    float r = -s[6];
    float wR = s[7];
    
    float V_fx = Vx;
    float V_fy = Vy+r*lf;
    float V_rx = Vx;
    float V_ry = Vy-r*lr;

    float VFx = V_fx*cosf(delta)+V_fy*sinf(delta);
    float VFy = -V_fx*sinf(delta)+V_fy*cosf(delta);
    float VRx = V_rx;
    float VRy = V_ry;

//    std::cout<<para;
 
    if (wR*r_R<0.05){ 
       s_der[3] = 0;
       s_der[5] = 0;
       s_der[6] = 0;
       s_der[7] = Tr/Iwr;
       s_der[4] = 0.8*s_der[7]*r_R;
	}
    else {
       float sFx = 0;
       float sFy = VFy/VFx;
       float sF = sqrtf( sFx*sFx+sFy*sFy );
       float sRx = ( VRx-wR*r_R )/( wR*r_R );
       float sRy = VRy/( wR*r_R ); 
       float sR = sqrtf( sRx*sRx+sRy*sRy );
       float mu_sF=D*sinf( C*atanf( B*( (1-E)*(sF+Sh)+E/B*atanf(B*(sF+Sh)) ) ) ) +Sv;
       float mu_sR=D*sinf( C*atanf( B*( (1-E)*(sR+Sh)+E/B*atanf(B*(sR+Sh)) ) ) ) +Sv;
       
       float fFx,fFy,fRx,fRy;
       if (sF==0){
	  fFx=0;
	  fFy=0; 
	     
	  float muRx=-sRx*mu_sR/sR;
	  float muRy=-sRy*mu_sR/sR;

          float fFz=( lr-h*muRx )*m*g/( L_axis-h*muRx );
          float fRz=m*g-fFz;
 
          fRx=muRx*fRz;
          fRy=muRy*fRz;      
        }
       else {
	  float muFx=-sFx*mu_sF/sF;
	  float muFy=-sFy*mu_sF/sF;  
	  float muRx=-sRx*mu_sR/sR;
	  float muRy=-sRy*mu_sR/sR; 

          float fFz=( lr-h*muRx )*m*g/( L_axis+h*( muFx*cosf( delta )-muFy*sinf( delta )- muRx) );
          float fRz=m*g-fFz;
 
          fFx=muFx*fFz;
          fFy=muFy*fFz;
          fRx=muRx*fRz;
          fRy=muRy*fRz;       
        }

        s_der[3] = 0;
	s_der[4] = ( fFx*cosf( delta )-fFy*sinf( delta )+fRx )/m+Vy*r;
        s_der[5] = ( fFx*sinf( delta )+fFy*cosf( delta )+fRy )/m-Vx*r;
        s_der[6] = -( ( fFy*cosf( delta )+fFx*sinf( delta ) )*lf-fRy*lr )/Iz;
        s_der[7] = (Tr-fRx*r_R)/Iwr;

        }

}

}
