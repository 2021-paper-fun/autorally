#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
 * @file mppi_adapitve_controller.cu
 * @author Jake Sacks <jsacks6@gatech.edu>
 * @date Jan 16, 2019
 * @copyright 2019 Georgia Institute of Technology
 * @brief Implementation of the MPPI_ADAPTIVE_CONTROLLER class.
 ***********************************************/

#define BLOCKSIZE_WRX MPPIAdaptiveController<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>::BLOCKSIZE_WRX
#define STATE_DIM DYNAMICS_T::STATE_DIM
#define CONTROL_DIM DYNAMICS_T::CONTROL_DIM
#define NUM_ROLLOUTS MPPIAdaptiveController<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>::NUM_ROLLOUTS

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
__global__ void weightedReductionNormalKernel(float* states_d, float* U_d, float* du_d, float* nu_d,
                                        float normalizer, int num_timesteps)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;

  __shared__ float u_system[STATE_DIM*((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1)];
  int stride = BLOCKSIZE_WRX;

  float u[CONTROL_DIM];

  int i,j;
  for (i = 0; i < CONTROL_DIM; i++) {
    u[i] = 0;
  }

  for (j = 0; j < CONTROL_DIM; j++) {
    u_system[tdx*CONTROL_DIM + j] = 0;
  }
  __syncthreads();

  if (BLOCKSIZE_WRX*tdx < NUM_ROLLOUTS) {
    float weight = 0;
    for (i = 0; i < stride; i++) {
      if (stride*tdx + i < NUM_ROLLOUTS) {
        weight = states_d[stride*tdx + i]/normalizer;
        for (j = 0; j < CONTROL_DIM; j++) {
          u[j] = U_d[bdx*CONTROL_DIM + j];
          u[j] -= du_d[(stride*tdx + i)*(num_timesteps*CONTROL_DIM) + bdx*CONTROL_DIM + j];
          u_system[tdx*CONTROL_DIM + j] += weight*u[j];
        }
      }
    }
  }
  __syncthreads();
  if (tdx == 0 && bdx < num_timesteps) {
    for (i = 0; i < CONTROL_DIM; i++) {
      u[i] = 0;
    }
    for (i = 0; i < (NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1; i++) {
      for (j = 0; j < CONTROL_DIM; j++) {
        u[j] += u_system[CONTROL_DIM*i + j];
      }
    }
    for (i = 0; i < CONTROL_DIM; i++) {
      du_d[CONTROL_DIM*bdx + i] = u[i];
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
__global__ void weightedReductionLaplaceKernel(float* states_d, float* U_d, float* du_d, float* nu_d,
                                               float normalizer, int num_timesteps)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;

  __shared__ float u_system[STATE_DIM*((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1)];
  int stride = BLOCKSIZE_WRX;

  float u[CONTROL_DIM];
  float du[CONTROL_DIM];

  int i,j;
  for (i = 0; i < CONTROL_DIM; i++) {
    u[i] = 0;
  }

  for (j = 0; j < CONTROL_DIM; j++) {
    u_system[tdx*CONTROL_DIM + j] = 0;
  }
  __syncthreads();

  float sign;
  if (BLOCKSIZE_WRX*tdx < NUM_ROLLOUTS) {
    float weight = 0;
    for (i = 0; i < stride; i++) {
      if (stride*tdx + i < NUM_ROLLOUTS) {
        weight = states_d[stride*tdx + i]/normalizer;
        for (j = 0; j < CONTROL_DIM; j++) {
          du[j] = du_d[(stride*tdx + i)*(num_timesteps*CONTROL_DIM) + bdx*CONTROL_DIM + j];
          u[j] =  U_d[bdx*CONTROL_DIM + j] - du[j];
          sign = u[j]>0 ? 1. : -1.;
          if (u[j] == 0) sign = 0.;
          u_system[tdx*CONTROL_DIM + j] += weight * nu_d[j] * sign;
        }
      }
    }
  }
  __syncthreads();
  if (tdx == 0 && bdx < num_timesteps) {
    for (i = 0; i < CONTROL_DIM; i++) {
      u[i] = 0;
    }
    for (i = 0; i < (NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1; i++) {
      for (j = 0; j < CONTROL_DIM; j++) {
        u[j] += u_system[CONTROL_DIM*i + j];
      }
    }
    for (i = 0; i < CONTROL_DIM; i++) {
      du_d[CONTROL_DIM*bdx + i] = u[i];
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
__global__ void weightedReductionCauchyKernel(float* states_d, float* U_d, float* du_d, float* nu_d,
                                              float normalizer, int num_timesteps)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;

  __shared__ float u_system[STATE_DIM*((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1)];
  int stride = BLOCKSIZE_WRX;

  float u[CONTROL_DIM];

  int i,j;
  for (i = 0; i < CONTROL_DIM; i++) {
    u[i] = 0;
  }

  for (j = 0; j < CONTROL_DIM; j++) {
    u_system[tdx*CONTROL_DIM + j] = 0;
  }
  __syncthreads();

  float sign;
  if (BLOCKSIZE_WRX*tdx < NUM_ROLLOUTS) {
    float weight = 0;
    for (i = 0; i < stride; i++) {
      if (stride*tdx + i < NUM_ROLLOUTS) {
        weight = states_d[stride*tdx + i]/normalizer;
        for (j = 0; j < CONTROL_DIM; j++) {
          u[j] = du_d[(stride*tdx + i)*(num_timesteps*CONTROL_DIM) + bdx*CONTROL_DIM + j];
          u[j] = U_d[bdx*CONTROL_DIM + j] - u[j];
          u_system[tdx*CONTROL_DIM + j] += weight * 4 * u[j] / (1 + pow(u[j]/nu_d[j], 2));
        }
      }
    }
  }
  __syncthreads();
  if (tdx == 0 && bdx < num_timesteps) {
    for (i = 0; i < CONTROL_DIM; i++) {
      u[i] = 0;
    }
    for (i = 0; i < (NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1; i++) {
      for (j = 0; j < CONTROL_DIM; j++) {
        u[j] += u_system[CONTROL_DIM*i + j];
      }
    }
    for (i = 0; i < CONTROL_DIM; i++) {
      du_d[CONTROL_DIM*bdx + i] = u[i];
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void launchWeightedReductionNormalKernel(float* state_costs_d, float* U_d, float* du_d, float* nu_d,
                                   float normalizer, int num_timesteps, hipStream_t stream)
{
  dim3 dimBlock((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1, 1, 1);
  dim3 dimGrid(num_timesteps, 1, 1);
  weightedReductionNormalKernel<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y><<<dimGrid, dimBlock, 0,
  stream>>> (state_costs_d, U_d, du_d, nu_d, normalizer, num_timesteps);
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void launchWeightedReductionLaplaceKernel(float* state_costs_d, float* U_d, float* du_d, float* nu_d,
                                   float normalizer, int num_timesteps, hipStream_t stream)
{
  dim3 dimBlock((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1, 1, 1);
  dim3 dimGrid(num_timesteps, 1, 1);
  weightedReductionLaplaceKernel<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y><<<dimGrid, dimBlock, 0, stream>>>
    (state_costs_d, U_d, du_d, nu_d, normalizer, num_timesteps);
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void launchWeightedReductionCauchyKernel(float* state_costs_d, float* U_d, float* du_d, float* nu_d,
                                          float normalizer, int num_timesteps, hipStream_t stream)
{
  dim3 dimBlock((NUM_ROLLOUTS-1)/BLOCKSIZE_WRX + 1, 1, 1);
  dim3 dimGrid(num_timesteps, 1, 1);
  weightedReductionCauchyKernel<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y><<<dimGrid, dimBlock, 0, stream>>>
    (state_costs_d, U_d, du_d, nu_d, normalizer, num_timesteps);
}

#undef BLOCKSIZE_WRX
#undef STATE_DIM
#undef CONTROL_DIM
#undef NUM_ROLLOUTS

/******************************************************************************************************************
MPPI Adaptive Controller implementation
*******************************************************************************************************************/

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
MPPIAdaptiveController<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>::MPPIAdaptiveController(
  DYNAMICS_T* model, COSTS_T* costs, OPTIMIZER_T* optim, int num_timesteps, int hz, float gamma,
  float* exploration_var, float* init_u, int num_optimization_iters, int opt_stride,
  hipStream_t stream, std::string dist_type)
  : MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>(model, costs, num_timesteps, hz, gamma,
       exploration_var, init_u, num_optimization_iters, opt_stride, stream), rng_(rd_())
{
  //Set the optimizer
  optim_ = optim;

  //Initialize vectors
  typedef MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y> Base;
  std_.assign(exploration_var, exploration_var + CONTROL_DIM);
  grads_.assign(Base::numTimesteps_*CONTROL_DIM, 0);
  noise_.assign(NUM_ROLLOUTS*Base::numTimesteps_*CONTROL_DIM, 0);

  //Set distribution type
  if (dist_type == "normal") {
    dist_type_ = NORMAL;
    for (int i=0; i<CONTROL_DIM; i++)
      norm_dist_.push_back(std::normal_distribution<float>(0., exploration_var[i]));
  } else if (dist_type == "laplace") {
    dist_type_ = LAPLACE;
    for (int i=0; i<CONTROL_DIM; i++) {
      exp_dist_.push_back(std::exponential_distribution<float>(1. / exploration_var[i]));
    }
  } else {
    dist_type_ = CAUCHY;
    for (int i=0; i<CONTROL_DIM; i++) {
      cauchy_dist_.push_back(std::cauchy_distribution<float>(0., exploration_var[i]));
    }
  }
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
MPPIAdaptiveController<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>::~MPPIAdaptiveController()
{
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIAdaptiveController<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>::slideControlSeq(int stride)
{
  MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>::slideControlSeq(stride);
  optim_->slideRunningEstimates(stride, CONTROL_DIM);
}

template<class DYNAMICS_T, class COSTS_T, class OPTIMIZER_T, int ROLLOUTS, int BDIM_X, int BDIM_Y>
void MPPIAdaptiveController<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>::computeControl(Eigen::Matrix<float, STATE_DIM, 1> state)
{
  typedef MPPIController<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y> Base;
  //First transfer the state and current control sequence to the device.
  Base::costs_->paramsToDevice();
  Base::model_->paramsToDevice();

  HANDLE_ERROR( hipMemcpyAsync(Base::state_d_, state.data(), STATE_DIM*sizeof(float), hipMemcpyHostToDevice, Base::stream_));
  for (int opt_iter = 0; opt_iter < Base::num_iters_; opt_iter++) {
    HANDLE_ERROR( hipMemcpyAsync(Base::U_d_, Base::U_.data(), CONTROL_DIM*Base::numTimesteps_*sizeof(float), hipMemcpyHostToDevice, Base::stream_));
    //Generate a bunch of random numbers
    for (int i=0; i<NUM_ROLLOUTS*Base::numTimesteps_; i++) {
      for (int j=0; j<CONTROL_DIM; j++) {
        if (dist_type_ == NORMAL) {
          noise_[i*CONTROL_DIM + j] = norm_dist_[j](rng_);
          //ROS_INFO("du: %f", noise_[i*CONTROL_DIM + j]);
        } else if (dist_type_ == LAPLACE) {
          noise_[i*CONTROL_DIM + j] = exp_dist_[j](rng_) - exp_dist_[j](rng_);
          //ROS_INFO("du: %f", noise_[i*CONTROL_DIM + j]);
        } else {
          noise_[i*CONTROL_DIM + j] = cauchy_dist_[j](rng_);
          //ROS_INFO("du: %f", noise_[i*CONTROL_DIM + j]);
        }
      }
    }

    HANDLE_ERROR(hipMemcpyAsync(Base::du_d_, noise_.data(),
                                 NUM_ROLLOUTS*Base::numTimesteps_*CONTROL_DIM*sizeof(float),
                                 hipMemcpyHostToDevice, Base::stream_));

    //hiprandGenerateNormal(Base::gen_, Base::du_d_, NUM_ROLLOUTS*Base::numTimesteps_*CONTROL_DIM, 0.0, 1.0);

    //Launch the rollout kernel
    launchRolloutKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>(Base::numTimesteps_, Base::state_d_, Base::U_d_,
                                                                       Base::du_d_, Base::nu_d_, Base::traj_costs_d_, Base::model_,
                                                                       Base::costs_, Base::optimizationStride_, Base::stream_);

    HANDLE_ERROR(hipMemcpyAsync(Base::traj_costs_.data(), Base::traj_costs_d_, NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost, Base::stream_));
    //NOTE: The calls to hipMemcpyAsync are only asynchronous with regards to (1) CPU operations AND (2) GPU operations
    //that are potentially occuring on other streams. Since all the previous kernel/memcpy operations use the same
    //stream, they all occur sequentially with respect to our stream (which is necessary for correct execution)

    //Synchronize stream here since we want to do computations on the CPU
    HANDLE_ERROR( hipStreamSynchronize(Base::stream_) );

    //Compute the baseline (minimum) sampled cost
    float baseline = Base::traj_costs_[0];
    for (int i = 0; i < NUM_ROLLOUTS; i++) {
      if (Base::traj_costs_[i] < baseline){
        baseline = Base::traj_costs_[i];
      }
    }

    //Now resume GPU computations
    launchNormExpKernel<DYNAMICS_T, COSTS_T, ROLLOUTS, BDIM_X, BDIM_Y>(Base::traj_costs_d_, Base::gamma_, baseline, Base::stream_);
    HANDLE_ERROR(hipMemcpyAsync(Base::traj_costs_.data(), Base::traj_costs_d_, NUM_ROLLOUTS*sizeof(float), hipMemcpyDeviceToHost, Base::stream_));
    hipStreamSynchronize(Base::stream_);

    //Compute the normalizing term
    Base::normalizer_ = 0;
    for (int i = 0; i < NUM_ROLLOUTS; i++) {
      Base::normalizer_ += Base::traj_costs_[i];
    }

    //Compute the cost weighted average.
     if (dist_type_ == NORMAL) {
       launchWeightedReductionNormalKernel<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>(
         Base::traj_costs_d_, Base::U_d_, Base::du_d_, Base::nu_d_, Base::normalizer_, Base::numTimesteps_,
         Base::stream_);
     } else if (dist_type_ == LAPLACE) {
       launchWeightedReductionLaplaceKernel<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>(
         Base::traj_costs_d_, Base::U_d_, Base::du_d_, Base::nu_d_, Base::normalizer_, Base::numTimesteps_,
         Base::stream_);
     } else if (dist_type_ == CAUCHY) {
       launchWeightedReductionCauchyKernel<DYNAMICS_T, COSTS_T, OPTIMIZER_T, ROLLOUTS, BDIM_X, BDIM_Y>(
         Base::traj_costs_d_, Base::U_d_, Base::du_d_, Base::nu_d_, Base::normalizer_, Base::numTimesteps_,
         Base::stream_);
     }
    //Transfer control update to host.
    HANDLE_ERROR( hipMemcpyAsync(Base::du_.data(), Base::du_d_, Base::numTimesteps_*CONTROL_DIM*sizeof(float),
                                  hipMemcpyDeviceToHost, Base::stream_));
    hipStreamSynchronize(Base::stream_);

    optim_->step(Base::U_, Base::du_);
  }
  //Smooth for the next optimization round
  Base::savitskyGolay();
  //Compute the planned trajectory
  Base::computeNominalTraj(state);
}
